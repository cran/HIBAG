#include "hip/hip_runtime.h"
// ===========================================================
//     _/_/_/   _/_/_/  _/_/_/_/    _/_/_/_/  _/_/_/   _/_/_/
//      _/    _/       _/             _/    _/    _/   _/   _/
//     _/    _/       _/_/_/_/       _/    _/    _/   _/_/_/
//    _/    _/       _/             _/    _/    _/   _/
// _/_/_/   _/_/_/  _/_/_/_/_/     _/     _/_/_/   _/_/
// ===========================================================
//
// gpuHLA.cpp: GPU supports for HLA Genotype Imputation
//
// Copyright (C) 2013	Xiuwen Zheng (zhengx@u.washington.edu)
//
// This file is part of HIBAG package.
//
// HIBAG is free software: you can redistribute it and/or modify it
// under the terms of the GNU General Public License Version 3 as
// published by the Free Software Foundation.
//
// HIBAG is distributed in the hope that it will be useful, but
// WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with HIBAG.
// If not, see <http://www.gnu.org/licenses/>.


// g++ -arch x86_64   -I/Developer/NVIDIA/CUDA-5.0/include -I. -I.. -I../../common/inc -o deviceQuery.o -c deviceQuery.cpp
// g++ -arch x86_64  -o deviceQuery deviceQuery.o -Xlinker -rpath /Developer/NVIDIA/CUDA-5.0/lib -L/Developer/NVIDIA/CUDA-5.0/lib -framework CUDA -lcudart 
// mkdir -p ../../bin/darwin/release
// cp deviceQuery ../../bin/darwin/release


// std::system includes
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <limits.h>
#include <StructHLA.h>
#include <math.h>

// CUDA-C includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>



// ---------------------------------------------------------------------------
// ---------------------------------------------------------------------------

typedef UINT8 TPackedHammingDistance[256u][256u];

__device__ static inline int _HamDist(int Length, UINT8 *SNP,
	UINT8 *H1, UINT8 *H2, TPackedHammingDistance *_PackedHamDist)
{
	int rv = 0;

	for (; Length >= 4; Length -= 4)  // one byte
	{
		rv += (*_PackedHamDist)[*SNP++][(*H1++) | (*H2++ << 1)];
	}
	if (Length > 0)
	{
		UINT8 mask = ~(0xFF << (((UINT8)Length)*2));
		rv += (*_PackedHamDist)[*SNP & mask][((*H1) | (*H2 << 1)) & mask];
	}

	return rv;
}

template<typename TFLOAT, typename TFLOAT_HAPLO>
__global__ void kernal_OutOfBagAcc_F32(int nHLA, int *_HLA_HapIdx, int nHaplo,
	TFLOAT_HAPLO *_HapList, int nGeno, TGPU_Genotype *_GList,
	int nSNP, UINT8 *_RetVal,
	TPackedHammingDistance *_PackedHamDist, TFLOAT *_RareFreq)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < nGeno)
	{
		_GList += idx;

		struct THLAType Guess;
		Guess.Allele1 = Guess.Allele2 = INT_MIN;

		TFLOAT_HAPLO *i1, *i2;
		TFLOAT max=0, prob;

		for (int h1=0; h1 < nHLA; h1++)
		{
			TFLOAT_HAPLO *L1_begin = _HapList + _HLA_HapIdx[h1*2];
			TFLOAT_HAPLO *L1_end   = _HapList + _HLA_HapIdx[h1*2] +
				_HLA_HapIdx[h1*2 + 1];

			// diag value
			prob = 0;
			for (i1=L1_begin; i1 != L1_end; i1++)
			{
				for (i2=i1; i2 != L1_end; i2++)
				{
					prob += ((i1 != i2) ? (2 * i1->Frequency * i2->Frequency) :
							(i1->Frequency * i2->Frequency)) *
						_RareFreq[_HamDist(nSNP, _GList->PackedSNPs,
							i1->PackedHaplo, i2->PackedHaplo, _PackedHamDist)];
				}
			}
			if (max < prob)
			{
				max = prob;
				Guess.Allele1 = Guess.Allele2 = h1;
			}

			// off-diag value
			for (int h2=h1+1; h2 < nHLA; h2++)
			{
				TFLOAT_HAPLO *L2_begin = _HapList + _HLA_HapIdx[h2*2];
				TFLOAT_HAPLO *L2_end   = _HapList + _HLA_HapIdx[h2*2] +
					_HLA_HapIdx[h2*2 + 1];

				prob = 0;
				for (i1=L1_begin; i1 != L1_end; i1++)
				{
					for (i2=L2_begin; i2 != L2_end; i2++)
					{
						prob += (2 * i1->Frequency * i2->Frequency) *
							_RareFreq[_HamDist(nSNP, _GList->PackedSNPs,
							i1->PackedHaplo, i2->PackedHaplo, _PackedHamDist)];
					}
				}
				if (max < prob)
				{
					max = prob;
					Guess.Allele1 = h1; Guess.Allele2 = h2;
				}
			}
		}

		// return 0, 1 or 2

		int T1 = _GList->HLA.Allele1, T2 = _GList->HLA.Allele2;
		int cnt = 0;
		if ((Guess.Allele1==T1) || (Guess.Allele1==T2))
		{
			cnt = 1;
			if (Guess.Allele1==T1) T1 = -1; else T2 = -1;
		}
		if ((Guess.Allele2==T1) || (Guess.Allele2==T2)) cnt ++;

		_RetVal[idx] = cnt;
	}
}


// C
extern "C"
{

static const char *ErrGPUMeg = "GPU fails!";

#define CUDA_MALLOC(ptr, size, cmd)	\
	err = hipMalloc((void**)&ptr, size); \
	if (err != hipSuccess) \
	{ \
		fprintf(stderr, "hipMalloc returned %s\n", hipGetErrorString(err)); \
		hlaGPU_Finalize(); \
		cmd; \
		throw ErrGPUMeg; \
	}

#define CUDA_MEM_CPY(d, s, size, type, cmd)	\
	err = hipMemcpy(d, s, size, type); \
	if (err != hipSuccess) \
	{ \
		fprintf(stderr, "hipMemcpy returned %s\n", hipGetErrorString(err)); \
		hlaGPU_Finalize(); \
		cmd; \
		throw ErrGPUMeg; \
	}



// ---------------------------------------------------------------------------
// ---------------------------------------------------------------------------

static TPackedHammingDistance *Dev_PackedHammingDistance = NULL;

static double *Dev_RareFreq_Float64 = NULL;
static float  *Dev_RareFreq_Float32 = NULL;

void hlaGPU_Finalize();


/// Export: initialize GPU computing library for HLA imputation
/// throw an error if fails
void hlaGPU_Initialize(uint8_t *hamdist)
{
	hipError_t err;

	// ***********************************************************************
	// Hamming Distance
	CUDA_MALLOC(Dev_PackedHammingDistance, 256*256, hlaGPU_Finalize());
	CUDA_MEM_CPY(Dev_PackedHammingDistance, hamdist, 256*256,
		hipMemcpyHostToDevice, hlaGPU_Finalize());


	// ***********************************************************************
	// tables for rare frequencies
	static const double MIN_RARE_FREQ_F64 = 1e-5;
	static const float  MIN_RARE_FREQ_F32 = 1e-5;
	double _RareFreq_Float64[HIBAG_MAXNUM_SNP_IN_CLASSIFIER*2];
	float  _RareFreq_Float32[HIBAG_MAXNUM_SNP_IN_CLASSIFIER*2];

	for (int i=0; i < HIBAG_MAXNUM_SNP_IN_CLASSIFIER*2; i++)
	{
		_RareFreq_Float64[i] = exp (i * log (MIN_RARE_FREQ_F64));
		_RareFreq_Float32[i] = expf(i * logf(MIN_RARE_FREQ_F32));
	}
	_RareFreq_Float64[0] = 1;
	_RareFreq_Float32[0] = 1;

	CUDA_MALLOC(Dev_RareFreq_Float64, sizeof(_RareFreq_Float64),
		hlaGPU_Finalize());
	CUDA_MEM_CPY(Dev_RareFreq_Float64, _RareFreq_Float64,
		sizeof(_RareFreq_Float64), hipMemcpyHostToDevice, hlaGPU_Finalize());
	CUDA_MALLOC(Dev_RareFreq_Float32, sizeof(_RareFreq_Float32),
		hlaGPU_Finalize());
	CUDA_MEM_CPY(Dev_RareFreq_Float32, _RareFreq_Float32,
		sizeof(_RareFreq_Float32), hipMemcpyHostToDevice, hlaGPU_Finalize());
}



/// Export: initialize GPU computing library for HLA imputation
void hlaGPU_Finalize()
{
	if (Dev_PackedHammingDistance)
	{
		hipFree(Dev_PackedHammingDistance);
		Dev_PackedHammingDistance = NULL;
	}
	if (Dev_RareFreq_Float64)
	{
		hipFree(Dev_RareFreq_Float64);
		Dev_RareFreq_Float64 = NULL;
	}
	if (Dev_RareFreq_Float32)
	{
		hipFree(Dev_RareFreq_Float32);
		Dev_RareFreq_Float32 = NULL;
	}
}



// ---------------------------------------------------------------------------
// ---------------------------------------------------------------------------

int hlaGPU_OutOfBagAcc_F32(int nHLA, int *_HLA_HapIdx, int nHaplo,
	TGPU_Haplotype_F32 *_HapList, int nGeno, TGPU_Genotype *_GList, int nSNP)
{
	if (nGeno <= 0) return 0;

	// copy host memory to device memory
	hipError_t err;

	int *Dev_HLA_HapIdx = NULL;
	TGPU_Haplotype_F32 *Dev_HapList = NULL;
	TGPU_Genotype *Dev_GList = NULL;
	uint8_t *Dev_RetVal = NULL;

	// allocate memory
	CUDA_MALLOC(Dev_HLA_HapIdx, sizeof(int)*nHLA*2, {});
	CUDA_MALLOC(Dev_HapList, sizeof(TGPU_Haplotype_F32)*nHaplo,
		hipFree(Dev_HLA_HapIdx));
	CUDA_MALLOC(Dev_GList, sizeof(TGPU_Genotype)*nGeno,
		{ hipFree(Dev_HLA_HapIdx); hipFree(Dev_HapList); });
	CUDA_MALLOC(Dev_RetVal, sizeof(uint8_t)*nGeno,
		{ hipFree(Dev_HLA_HapIdx); hipFree(Dev_HapList); hipFree(Dev_GList); });

	// memory copy
	CUDA_MEM_CPY(Dev_HLA_HapIdx, _HLA_HapIdx, sizeof(int)*nHLA*2,
		hipMemcpyHostToDevice,  { hipFree(Dev_HLA_HapIdx);
		hipFree(Dev_HapList); hipFree(Dev_GList); hipFree(Dev_RetVal); });
	CUDA_MEM_CPY(Dev_HapList, _HapList, sizeof(TGPU_Haplotype_F32)*nHaplo,
		hipMemcpyHostToDevice,  { hipFree(Dev_HLA_HapIdx);
		hipFree(Dev_HapList); hipFree(Dev_GList); hipFree(Dev_RetVal); });
	CUDA_MEM_CPY(Dev_GList, _GList, sizeof(TGPU_Genotype)*nGeno,
		hipMemcpyHostToDevice,  { hipFree(Dev_HLA_HapIdx);
		hipFree(Dev_HapList); hipFree(Dev_GList); hipFree(Dev_RetVal); });

	// run in parallel
	dim3 dimBlock(32);
	dim3 dimGrid((nGeno/32) + ((nGeno % 32) ? 1:0));

	kernal_OutOfBagAcc_F32<float, TGPU_Haplotype_F32><<<dimGrid, dimBlock>>>(
		nHLA, Dev_HLA_HapIdx, nHaplo, Dev_HapList, nGeno, Dev_GList, nSNP,
		Dev_RetVal, Dev_PackedHammingDistance, Dev_RareFreq_Float32);


	// merge results
	int rv = 0;
	if (nGeno <= 16384)
	{
		UINT8 buffer[16384];
		CUDA_MEM_CPY(buffer, Dev_RetVal, nGeno,
			hipMemcpyDeviceToHost,  { hipFree(Dev_HLA_HapIdx);
			hipFree(Dev_HapList); hipFree(Dev_GList); hipFree(Dev_RetVal); });
		for (int i=0; i < nGeno; i++) rv += buffer[i];
	} else {
		UINT8 *buffer = new UINT8[nGeno];
		CUDA_MEM_CPY(buffer, Dev_RetVal, nGeno,
			hipMemcpyDeviceToHost,  { hipFree(Dev_HLA_HapIdx);
			hipFree(Dev_HapList); hipFree(Dev_GList); hipFree(Dev_RetVal);
			delete []buffer; });
		for (int i=0; i < nGeno; i++) rv += buffer[i];
		delete []buffer;
	}

	hipFree(Dev_HLA_HapIdx);
	hipFree(Dev_HapList);
	hipFree(Dev_GList);
	hipFree(Dev_RetVal);

	return rv;
}


int hlaGPU_OutOfBagAcc_F64(int nHLA, int *_HLA_HapIdx,
	TGPU_Haplotype_F64 *_HapList, int nGeno, TGPU_Genotype *_GList)
{
	return 0;	
}


// ---------------------------------------------------------------------------
// ---------------------------------------------------------------------------

#if CUDART_VERSION < 5000

/// This function wraps the CUDA Driver API into a template function
static inline void getCudaAttribute(int *attribute,
	hipDeviceAttribute_t device_attribute, int device)
{
	hipError_t error = hipDeviceGetAttribute(attribute, device_attribute, device);
	if (hipSuccess != error)
	{
		fprintf(stderr,
			"cuSafeCallNoSync() Driver API error = %04d from file <%s>, line %i.\n",
			error, __FILE__, __LINE__);
		throw ErrGPUMeg;
	}
}

#endif


/// Export: enumerate the properties of the CUDA devices present in the system
/// throw an error if fails
void hlaGPU_DeviceQuery()
{
	printf("CUDA Device Query:\n\n");

	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess)
	{
		fprintf(stderr, "hipGetDeviceCount returned %d\n-> %s\n",
			(int)error_id, hipGetErrorString(error_id));
		throw ErrGPUMeg;
	}

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount <= 0)
	{
		fprintf(stderr, "There are no available device(s) that support CUDA\n");
		throw ErrGPUMeg;
	} else {
		printf("Detected %d CUDA Capable device(s)\n", deviceCount);
	}

	int dev, driverVersion = 0, runtimeVersion = 0;
	for (dev = 0; dev < deviceCount; dev++)
    {
		hipSetDevice(dev);
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);

		printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

		// Console log
		hipDriverGetVersion(&driverVersion);
		hipRuntimeGetVersion(&runtimeVersion);
		printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
			driverVersion/1000, (driverVersion%100)/10,
			runtimeVersion/1000, (runtimeVersion%100)/10);
		printf("  CUDA Capability Major/Minor version number:    %d.%d\n",
			deviceProp.major, deviceProp.minor);

		printf("  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
			(float)deviceProp.totalGlobalMem/1048576.0f,
			(unsigned long long) deviceProp.totalGlobalMem);

		printf("  (%2d) Multiprocessors x (%3d) CUDA Cores/MP:    %d CUDA Cores\n",
			deviceProp.multiProcessorCount,
			_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
			_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) *
			deviceProp.multiProcessorCount);
		printf("  GPU Clock rate:                                %.0f MHz (%0.2f GHz)\n",
			deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);

#if CUDART_VERSION >= 5000
		// This is supported in CUDA 5.0 (runtime API device properties)
		printf("  Memory Clock rate:                             %.0f Mhz\n",
			deviceProp.memoryClockRate * 1e-3f);
		printf("  Memory Bus Width:                              %d-bit\n",
			deviceProp.memoryBusWidth);

		if (deviceProp.l2CacheSize)
		{
			printf("  L2 Cache Size:                                 %d bytes\n",
				deviceProp.l2CacheSize);
		}
#else
		// This only available in CUDA 4.0-4.2 (but these were only exposed in the CUDA Driver API)
		int memoryClock;
		getCudaAttribute(&memoryClock, hipDeviceAttributeMemoryClockRate, dev);
		printf("  Memory Clock rate:                             %.0f Mhz\n",
			memoryClock * 1e-3f);
		int memBusWidth;
		getCudaAttribute(&memBusWidth, hipDeviceAttributeMemoryBusWidth, dev);
		printf("  Memory Bus Width:                              %d-bit\n",
			memBusWidth);
		int L2CacheSize;
		getCudaAttribute(&L2CacheSize, hipDeviceAttributeL2CacheSize, dev);

		if (L2CacheSize)
		{
			printf("  L2 Cache Size:                                 %d bytes\n",
				L2CacheSize);
		}
#endif

		printf("  Max Texture Dimension Size (x,y,z)             1D=(%d), 2D=(%d,%d), 3D=(%d,%d,%d)\n",
			deviceProp.maxTexture1D   , deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1],
			deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
        printf("  Max Layered Texture Size (dim) x layers        1D=(%d) x %d, 2D=(%d,%d) x %d\n",
			deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1],
			deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1],
			deviceProp.maxTexture2DLayered[2]);

		printf("  Total amount of constant memory:               %lu bytes\n",
			deviceProp.totalConstMem);
		printf("  Total amount of shared memory per block:       %lu bytes\n",
			deviceProp.sharedMemPerBlock);
		printf("  Total number of registers available per block: %d\n",
			deviceProp.regsPerBlock);
		printf("  Warp size:                                     %d\n",
			deviceProp.warpSize);
		printf("  Maximum number of threads per multiprocessor:  %d\n",
			deviceProp.maxThreadsPerMultiProcessor);
		printf("  Maximum number of threads per block:           %d\n",
			deviceProp.maxThreadsPerBlock);
		printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
			deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
			deviceProp.maxThreadsDim[2]);
		printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
			deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
			deviceProp.maxGridSize[2]);
		printf("  Maximum memory pitch:                          %lu bytes\n",
			deviceProp.memPitch);
		printf("  Texture alignment:                             %lu bytes\n",
			deviceProp.textureAlignment);
		printf("  Concurrent copy and kernel execution:          %s with %d copy engine(s)\n",
			(deviceProp.deviceOverlap ? "Yes" : "No"), deviceProp.asyncEngineCount);
		printf("  Run time limit on kernels:                     %s\n",
			deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
		printf("  Integrated GPU sharing Host Memory:            %s\n",
			deviceProp.integrated ? "Yes" : "No");
		printf("  Support host page-locked memory mapping:       %s\n",
			deviceProp.canMapHostMemory ? "Yes" : "No");
		printf("  Alignment requirement for Surfaces:            %s\n",
			deviceProp.surfaceAlignment ? "Yes" : "No");
		printf("  Device has ECC support:                        %s\n",
			deviceProp.ECCEnabled ? "Enabled" : "Disabled");
#ifdef WIN32
		printf("  CUDA Device Driver Mode (TCC or WDDM):         %s\n",
			deviceProp.tccDriver ?
			"TCC (Tesla Compute Cluster Driver)" : "WDDM (Windows Display Driver Model)");
#endif
		printf("  Device supports Unified Addressing (UVA):      %s\n",
			deviceProp.unifiedAddressing ? "Yes" : "No");
		printf("  Device PCI Bus ID / PCI location ID:           %d / %d\n",
			deviceProp.pciBusID, deviceProp.pciDeviceID);

		const static char *sComputeMode[] =
		{
			"Default (multiple host threads can use ::hipSetDevice() with device simultaneously)",
			"Exclusive (only one host thread in one process is able to use ::hipSetDevice() with this device)",
			"Prohibited (no host thread can use ::hipSetDevice() with this device)",
			"Exclusive Process (many threads in one process is able to use ::hipSetDevice() with this device)",
			"Unknown",
			NULL
		};
		printf("  Compute Mode:\n");
		printf("     < %s >\n", sComputeMode[deviceProp.computeMode]);
	}

	// csv masterlog info
	// *****************************
	// exe and CUDA driver name
	printf("\n");
	printf("deviceQuery, CUDA Driver = CUDART");

	// driver version
	printf(", CUDA Driver Version = %d.%d",
		driverVersion/1000, (driverVersion%100)/10);
	// Runtime version
	printf(", CUDA Runtime Version = %d.%d",
		runtimeVersion/1000, (runtimeVersion%100)/10);
	// Device count
	printf(", NumDevs = %d", deviceCount);

	// Print Out all device Names
	for (dev = 0; dev < deviceCount; ++dev)
	{
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);
		printf(", Device%d = %s", dev, deviceProp.name);
	}

	printf("\n");
}


} // extern "C"
